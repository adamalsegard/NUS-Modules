/** 
 * 
 * Matrix Multiplication - CUDA for GPUs
 *
 * CS3210
 *
 * Modified by Adam Alsegard, A0153097B, Assignment 1
 * Optimization v.2 - best version!
 * 
 **/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#include <assert.h>

int size;
// Use block size 32 because of warp size (32), maximum threads per block: 1024 (32x32)
#define BLOCK_SIZE 32

typedef struct
{
	float ** element;
} matrix;


long long wall_clock_time()
{
#ifdef __linux__
	struct timespec tp;
	clock_gettime(CLOCK_REALTIME, &tp);
	return (long long)(tp.tv_nsec + (long long)tp.tv_sec * 1000000000ll);
#else
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return (long long)(tv.tv_usec * 1000 + (long long)tv.tv_sec * 1000000000ll);
#endif
}

/**
 * Allocates memory for a matrix of size SIZE
 * The memory is allocated row-major order, i.e. 
 *  elements from the same row are allocated at contiguous 
 *  memory addresses.
 **/
__host__ void allocate_matrix(matrix* m)
{
	int i;
	hipError_t rc;
	
	// allocate array for all the rows
	rc = hipMallocManaged((void**)&(m->element), sizeof(float*) * size);
	if (rc != hipSuccess)
	{
		fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(rc));
		exit(1);
	}
	
	// allocate an array for each row of the matrix
	for (i = 0; i < size; i++)
	{
		rc = hipMallocManaged((void**)&(m->element[i]), sizeof(float) * size);
		if (rc != hipSuccess)
		{
			fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(rc));
			exit(1);
		}
	}
}

/**
 * Free the memory allocated for a matrix.
 **/
__host__ void free_matrix(matrix* m) {
	int i;
	for (i = 0; i < size; i++)
		hipFree(m->element[i]);
	hipFree(m->element);
}

/**
 * Initializes the elements of the matrix with
 * random values between 0 and 9
 **/
__host__ void init_matrix(matrix m)
{
	int i, j;
	
	for (i = 0; i < size; i++)
		for (j = 0; j < size; j++)
		{
			m.element[i][j] = rand() % 10;
		}
}

/**
 * Initializes the elements of the matrix with
 * element 0.
 **/
__host__ void init_matrix_zero(matrix m)
{
	int i, j;
	
	for (i = 0; i < size; i++)
		for (j = 0; j < size; j++)
		{
			m.element[i][j] = 0.0;
		}
}


/**
 * Multiplies matrix @a with matrix @b storing
 * the result in matrix @result
 * 
 * The multiplication algorithm is the O(n^3) 
 * algorithm
 */
void mm(matrix a, matrix b, matrix result)
{
	int i, j, k;
	
	// Do the multiplication
	for (i = 0; i < size; i++)
		for (j = 0; j < size; j++)
			for(k = 0; k < size; k++)
				result.element[i][j] += a.element[i][k] * b.element[k][j];
}

/**
 * Each kernel computes the result element (i,j).
 */
__global__ void mm_kernel(matrix a, matrix b, matrix result, int size)
{

	// Find out which row and column we are on
	int row = blockIdx.y * blockDim.y + threadIdx.y; 
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int newSize = size;

	// If the size is not a multiple of blkSize, add one iteration to the for-loop
	if ( newSize%BLOCK_SIZE != 0 )
	{
		newSize += BLOCK_SIZE;
	}
	
	// Each thread will calculate one value for the resulting matrix
	float tmpRes = 0;

	// Find out which thread we are on inside the block
	int threadRow = threadIdx.y;
	int threadCol = threadIdx.x;	
		
	// Gather all subMatrices we need and copy them to shared memory
	for(int i=0; i < newSize; i+=BLOCK_SIZE)
	{			
		
		__shared__ float shA[BLOCK_SIZE][BLOCK_SIZE];
		__shared__ float shB[BLOCK_SIZE][BLOCK_SIZE];

		// Blocks from A moves horizontally, B vertically. If we are outside of matrices, set to zero.
		if( row < size && (i+threadCol) < size)
	 		shA[threadRow][threadCol] = a.element[row][i+threadCol];		
		else
			shA[threadRow][threadCol] = 0;

		if (col < size && (i+threadRow) < size)
			shB[threadRow][threadCol] = b.element[i+threadRow][col];
		else
			shB[threadRow][threadCol] = 0;

		// Sync all threads (block level) to make sure all needed elements are loaded
        __syncthreads();

		// Calculate the new value
		for(int k=0; k < BLOCK_SIZE; k++)
			tmpRes += shA[threadRow][k] * shB[k][threadCol];

		// Sync threads again to make sure the computation is done before loading the next matrices
		__syncthreads();
		
	}

	// Every thread copies one value, ignore the zero values
	if(row < size && col < size)
		result.element[row][col] = tmpRes;
}

void print_matrix(matrix m)
{
	int i, j;
	
	for (i = 0; i < size; i++)
	{
		printf("row %4d: ", i);
		for (j = 0; j < size; j++)
			printf("%6.2f  ", m.element[i][j]);
		printf("\n");
	}
}



void work()
{
	matrix a, b, result1, result2;
	long long before, after;
	int correct, i, j, dim;
	hipError_t rc;

	// Allocate memory for matrices
	allocate_matrix(&a);
	allocate_matrix(&b);
	allocate_matrix(&result1);
	allocate_matrix(&result2);	

	// Initialize matrix elements
	init_matrix(a);
	init_matrix(b);

	// Perform sequential matrix multiplication
	before = wall_clock_time();
	mm(a, b, result1);
	after = wall_clock_time();
        fprintf(stderr, "Matrix multiplication on CPU took %1.2f seconds\n", ((float)(after - before))/1000000000);
		
	// Perform CUDA matrix  multiplication
	dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);	// a block of blkSize X blkSize CUDA threads
	dim = (size%BLOCK_SIZE==0) ? size/BLOCK_SIZE : size/BLOCK_SIZE + 1;
	dim3 blocksPerGrid(dim, dim);					// a grid of CUDA thread blocks, covers all elements in matrices
	before = wall_clock_time();
	mm_kernel<<<blocksPerGrid, threadsPerBlock>>>(a, b, result2, size);
	hipDeviceSynchronize();
	after = wall_clock_time();
	fprintf(stderr, "Matrix multiplication on GPU took %1.2f seconds\n", ((float)(after - before))/1000000000);

	// was there any error?
        rc = hipGetLastError();
        if (rc != hipSuccess)
                printf("Last CUDA error %s\n", hipGetErrorString(rc));

	// Compare the results
	correct = 1;
	for (i = 0; correct && i < size; i++)
		for (j = 0; j < size; j++)
			if (result1.element[i][j] != result2.element[i][j]) {
				correct = 0;
				break;
			}

	if (correct)
		printf("The result matrices are identical!\n");
	else
		printf("Difference in result matrices at element (%d, %d)!\n", i, j);

	free_matrix(&a);
	free_matrix(&b);
	free_matrix(&result1);
	free_matrix(&result2);
}


int main(int argc, char ** argv)
{
	srand(0); 

	printf("Usage: %s <size>\n", argv[0]);
    
	if (argc >= 2)
		size = atoi(argv[1]);
	else
		size = 1024;
		
	fprintf(stderr,"Sequential matrix multiplication of size %d\n", size);
    
	// Multiply the matrices
	work();

	return 0;
}
